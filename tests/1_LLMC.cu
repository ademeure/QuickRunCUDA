
#include <hip/hip_runtime.h>
// ----------------------------------------------------------------------------
// Packed128 data structure, which forces the compiler to use 128-bit loads/stores
// in GPUs that support (the LDG.128 and STS.128 instructions)
// This is a bit similar to the use of float4 in the case of 32-bit floats, but
// supports arbitrary precision.

template<class ElementType>
struct alignas(16) Packed128 {
    // Note: = default implicitly generates a __device__ function, but explicitly
    // adding __device__ causes a lot of warnings.
    Packed128() = default;
    __device__ explicit Packed128(int4 bits) {
        static_assert(sizeof(bits) == sizeof(payload), "Size mismatch.");
        memcpy(&payload, &bits, sizeof(bits));
    }

    __device__  static Packed128 constant(ElementType value) {
        Packed128 result;
        for(int k = 0; k < size; ++k) {
            result.payload[k] = value;
        }
        return result;
    }

    __device__ static Packed128 zeros() {
        return constant(0);
    }

    __device__ static Packed128 ones() {
        return constant(1);
    }

    __device__ ElementType& operator[](int index) {
        return payload[index];
    }
    __device__ const ElementType& operator[](int index) const {
        return payload[index];
    }
    __device__ int4 get_bits() const {
        int4 bits;
        static_assert(sizeof(bits) == sizeof(payload), "Size mismatch.");
        memcpy(&bits, &payload, sizeof(bits));
        return bits;
    }
    // e.g. sizeof(int4) is 16 (4 X 4 bytes), sizeof(bfloat16) = 2, so size = 8
    // so in the case where ElementType = bfloat16, we store 8 elements in one Packed128
    static constexpr const int size = sizeof(int4) / sizeof(ElementType);
    ElementType payload[size];
};

// short-form typedef
typedef Packed128<float> f128;

// load a Packed128 from an aligned memory address
template<class ElementType>
__device__ Packed128<ElementType> load128(const ElementType* address) {
    return Packed128<ElementType>{*reinterpret_cast<const int4*>(address)};
}
// load a Packed128 from an aligned memory address with streaming cache hint
template<class ElementType>
__device__ Packed128<ElementType> load128cs(const ElementType* address) {
    return Packed128<ElementType>{__ldcs(reinterpret_cast<const int4*>(address))};
}
// load a Packed128 from an aligned memory address with streaming cache hint
template<class ElementType>
__device__ Packed128<ElementType> load128cg(const ElementType* address) {
    return Packed128<ElementType>{__ldcg(reinterpret_cast<const int4*>(address))};
}
// store a Packed128 to an aligned memory address
template<class ElementType>
__device__ void store128(ElementType* target, Packed128<ElementType> value) {
    *reinterpret_cast<int4*>(target) = value.get_bits();
}
// store a Packed128 to an aligned memory address with streaming cache hint
template<class ElementType>
__device__ void store128cs(ElementType* target, Packed128<ElementType> value) {
    __stcs(reinterpret_cast<int4*>(target), value.get_bits());
}
// store a Packed128 to an aligned memory address while caching in L2 but bypassing L1
template<class ElementType>
__device__ void store128cg(ElementType* target, Packed128<ElementType> value) {
    __stcg(reinterpret_cast<int4*>(target), value.get_bits());
}

// ----------------------------------------------------------------------------
// reduced/mixed precision utilities

#if defined(ENABLE_BF16)

typedef __nv_bfloat16 floatX;
typedef __nv_bfloat16 floatN;
#define CUBLAS_LOWP CUDA_R_16BF // CUDA_R_16F or CUDA_R_16BF (or CUDA_R_32F)
// CUBLAS_COMPUTE_32F or CUBLAS_COMPUTE_16F (for CUDA_R_16F only, potentially slower?!)
#define CUBLAS_LOWP_COMPUTE CUBLAS_COMPUTE_32F

#elif defined(ENABLE_FP16)

typedef half floatX;
typedef half floatN;

#else

typedef float floatX;
typedef float floatN;
#endif

typedef Packed128<floatX> x128;

extern "C" __global__  void kernel(float *in, float* out, float* unused_C, int unused_0, int unused_1, int unused_2) {
    int id = (threadIdx.x + blockIdx.x * blockDim.x) * f128::size;

    f128 input = load128(in + id);
    for (int i = 0; i < f128::size; ++i) {
        input[i] = max(input[i], 0.0f); // RELU
    }
    store128(out + id, input);
}
